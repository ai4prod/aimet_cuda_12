#include "hip/hip_runtime.h"
//==============================================================================
//
//  @@-COPYRIGHT-START-@@
//
//  Copyright (c) 2016-2017, Qualcomm Innovation Center, Inc. All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software
//     without specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
//  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
//  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
//  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
//  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
//  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  @@-COPYRIGHT-END-@@
//
//==============================================================================

#include <stdexcept>

#include "cuda_util.hpp"
#include "trim_functions.cuh"
#include "trim_functions.hpp"

namespace DlQuantization
{
template <typename DTYPE>
__global__ void quantizeDequantizeKernel(const DTYPE* in, int cnt, DTYPE* out,
                                         DTYPE encoding_min, DTYPE encoding_max,
                                         DTYPE encoding_delta, DTYPE encoding_offset,
                                         RoundingMode rounding_mode)
{
    CUDA_KERNEL_LOOP(i, cnt)
    {
        quantizeToFxpDevice<DTYPE>(in + i, out + i,
                                   encoding_min, encoding_max,
                                   encoding_delta, encoding_offset,
                                   rounding_mode, i);
        dequantizeFromFxpDevice<DTYPE>(out + i, encoding_delta, encoding_offset);
    }
}

template <typename DTYPE>
__global__ void quantizeToFxpKernel(const DTYPE* in, int cnt, DTYPE* out,
                                    DTYPE encoding_min, DTYPE encoding_max,
                                    DTYPE encoding_delta, DTYPE encoding_offset,
                                    RoundingMode rounding_mode, unsigned int shift)
{
    CUDA_KERNEL_LOOP(i, cnt)
    {
        quantizeToFxpDevice<DTYPE>(in + i, out + i,
                                   encoding_min, encoding_max,
                                   encoding_delta, encoding_offset,
                                   rounding_mode, i);
        *(out + i) -= shift;
    }
}

template <typename DTYPE>
void quantizeDequantizeGpu(const DTYPE* in, int cnt, const TfEncoding& encoding,
                           DTYPE* out, RoundingMode rounding_mode)
{
    quantizeDequantizeKernel<DTYPE><<<CUDA_NUM_BLOCKS(cnt), CUDA_NUM_THREADS>>>(
            in, cnt, out, encoding.min, encoding.max, encoding.delta,
            encoding.offset, rounding_mode);
}

template <typename DTYPE>
void quantizeToFxpGpu(const DTYPE* in, int cnt, const TfEncoding& encoding,
                      DTYPE* out, RoundingMode rounding_mode, bool shiftToSigned)
{
    unsigned int shift = 0;
    if (shiftToSigned) {
        shift = pow(2, encoding.bw - 1);
    }
    quantizeToFxpKernel<DTYPE><<<CUDA_NUM_BLOCKS(cnt), CUDA_NUM_THREADS>>>(
            in, cnt, out, encoding.min, encoding.max, encoding.delta,
            encoding.offset, rounding_mode, shift);
}

// Explicit instantiations
template void quantizeDequantizeGpu(const double* in, int cnt, const TfEncoding& encoding, double* out,
                                    RoundingMode rounding_mode);

template void quantizeDequantizeGpu(const float* in, int cnt, const TfEncoding& encoding, float* out,
                                    RoundingMode rounding_mode);

template void quantizeToFxpGpu(const double* in, int cnt, const TfEncoding& encoding, double* out,
                               RoundingMode rounding_mode, bool shiftToSigned);


template void quantizeToFxpGpu(const float* in, int cnt, const TfEncoding& encoding, float* out,
                               RoundingMode rounding_mode, bool shiftToSigned);
}   // End of namespace DlQuantization
